#include "hip/hip_runtime.h"
#include <iostream>  //todo: fix issue in bght
#include <limits>
#include <stdlib.h>

#include "iht.hpp"
#include <cuco/static_multimap.cuh>

#include <thrust/transform.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>

#include <gunrock/formats/formats.hxx>   // formats (csr, coo)
#include <gunrock/memory.hxx>            // memory space
#include <gunrock/io/matrix_market.hxx>  // matrix_market support
#include <gunrock/util/timer.hxx>

using namespace gunrock;
using namespace memory;

int test_bucketing(int num_arguments, char** argument_array) {
  if (num_arguments != 3) {
    std::cerr << "usage: ./bin/<program-name> filename.mtx load-factor (0 to 1)"
              << std::endl;
    exit(1);
  }
  // --
  // Define types
  // Specify the types that will be used for
  // - vertex ids (vertex_t)
  // - edge offsets (edge_t)
  // - edge weights (weight_t)

  using vertex_t = uint32_t;
  using edge_t = uint32_t;
  using weight_t = float;

  using pair_type = bght::pair<vertex_t, edge_t>;
  using cuco_pair_type = cuco::pair_type<vertex_t, edge_t>;
  std::string filename = argument_array[1];
  float load_factor = std::atof(argument_array[2]);

  // Load the matrix-market dataset into csr format.
  // See `format` to see other supported formats.
  io::matrix_market_t<vertex_t, edge_t, weight_t> mm;
  using coo_t =
      format::coo_t<memory_space_t::device, vertex_t, edge_t, weight_t>;
  coo_t coo = mm.load(filename);

  auto n_edges = coo.number_of_nonzeros;

  std::cout << "Graph: " << filename << std::endl;
  std::cout << "Load factor: " << load_factor << std::endl;
  std::cout << "Number of edges: " << n_edges << std::endl;

  thrust::device_vector<pair_type> coo_as_pairs(n_edges);

  auto to_pair = [=] __device__(const auto& t) {
    return pair_type(thrust::get<0>(t), thrust::get<1>(t));
  };

  // todo: fix policy
  thrust::transform(thrust::device,
                    thrust::make_zip_iterator(thrust::make_tuple(
                        coo.row_indices.begin(), coo.column_indices.begin())),
                    thrust::make_zip_iterator(thrust::make_tuple(
                        coo.row_indices.end(), coo.column_indices.end())),
                    coo_as_pairs.begin(), to_pair);

  // hash table types
  using hash = bght::MurmurHash3_32<vertex_t>;
  using key_equal = bght::equal_to<vertex_t>;
  using allocator = bght::cuda_allocator<char>;
  static constexpr int bucket_size = 16;
  static constexpr int threshold = 14;
  using hash_map =
      bght::iht<vertex_t, vertex_t, hash, key_equal, cuda::thread_scope_device,
                allocator, bucket_size, threshold>;

  using cuco_map = cuco::static_multimap<vertex_t, vertex_t>;

  std::size_t capacity = static_cast<double>(n_edges) / load_factor;

  auto sentinel_key = std::numeric_limits<vertex_t>::max();
  auto sentinel_value = std::numeric_limits<vertex_t>::max();
  // hash_map map(capacity, cuco::sentinel::empty_key{sentinel_key},
  //              cuco::sentinel::empty_value{sentinel_value});
  hash_map map(capacity, sentinel_key, sentinel_value);
  util::timer_t timer;
  timer.begin();
  auto success{true};
  success = map.insert(coo_as_pairs.begin(), coo_as_pairs.end());
  auto elapsed = timer.end();
  if (success) {
    std::cout << "Rate: "
              << static_cast<double>(n_edges) / (elapsed * 0.001) * 1e-6
              << " MEdge/s" << std::endl;
    std::cout << "Elapsed: " << elapsed << " ms" << std::endl;
    std::cout << "succesuflly built the multimap with load factor = "
              << load_factor << std::endl;
    hipDeviceSynchronize();
  } else {
    std::cout << "failed to build the multimap with load factor = "
              << load_factor << std::endl;
  }
}

// Main method, wrapping test function
int main(int argc, char** argv) {
  test_bucketing(argc, argv);
}