#include "hip/hip_runtime.h"
#include <vector>

#include <gunrock/algorithms/tc.hxx>
#include <gunrock/util/timer.hxx>

#include <cxxopts.hpp>

using namespace gunrock;
using namespace memory;

#define cuda_try(call)                                        \
  do {                                                        \
    hipError_t err = call;                                   \
    if (err != hipSuccess) {                                 \
      printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__, \
             hipGetErrorString(err));                        \
      std::terminate();                                       \
    }                                                         \
  } while (0)

template <class T>
struct managed_allocator {
  typedef std::size_t size_type;
  typedef std::ptrdiff_t difference_type;

  typedef T value_type;
  typedef T* pointer;
  typedef const T* const_pointer;
  typedef T& reference;
  typedef const T& const_reference;

  template <class U>
  struct rebind {
    typedef managed_allocator<U> other;
  };
  managed_allocator() = default;
  template <class U>
  constexpr managed_allocator(const managed_allocator<U>&) noexcept {}
  T* allocate(std::size_t n) {
    void* p = nullptr;
    cuda_try(hipMallocManaged(&p, n * sizeof(T)));
    return static_cast<T*>(p);
  }
  void deallocate(T* p, std::size_t) noexcept { cuda_try(hipFree(p)); }
};

struct parameters_t {
  std::string filename;
  cxxopts::Options options;
  bool validate;
  bool reduce_all_triangles;

  /**
   * @brief Construct a new parameters object and parse command line arguments.
   *
   * @param argc Number of command line arguments.
   * @param argv Command line arguments.
   */
  parameters_t(int argc, char** argv)
      : options(argv[0], "Traingle Counting example") {
    // Add command line options
    options.add_options()("help", "Print help")(
        "validate", "CPU validation",
        cxxopts::value<bool>()->default_value("false"))(
        "m,market", "Matrix file", cxxopts::value<std::string>())(
        "r,reduce",
        "Compute a single triangle count for the entire graph (default = "
        "false)",
        cxxopts::value<bool>()->default_value("false"));

    // Parse command line arguments
    auto result = options.parse(argc, argv);

    if (result.count("help") || (result.count("market") == 0)) {
      std::cout << options.help({""}) << std::endl;
      std::exit(0);
    }
    filename = result["market"].as<std::string>();
    validate = result["validate"].as<bool>();
    reduce_all_triangles = result["reduce"].as<bool>();
  }
};

__global__ void average_neighbors(uint32_t* row_offsets,
                                  uint32_t* column_indices,
                                  uint32_t n_vertices,
                                  uint32_t* output,
                                  uint32_t gpu_offset_start,
                                  uint32_t gpu_offset_end) {
  auto vertex_index = threadIdx.x + blockIdx.x * blockDim.x;
  vertex_index += gpu_offset_start;

  if (vertex_index < gpu_offset_end) {
    auto row_start = row_offsets[vertex_index];
    auto row_end = row_offsets[vertex_index + 1];
    auto n_neighbors = row_end - row_start;

    uint32_t sum = 0;
    for (uint32_t n = 0; n < n_neighbors; n++) {
      sum += column_indices[row_start + n];
    }
    output[vertex_index] = sum;
  }
}

void test_tc(int num_arguments, char** argument_array) {
  // --
  // Define types

  using vertex_t = uint32_t;
  using edge_t = uint32_t;
  using weight_t = float;
  using count_t = vertex_t;

  using csr_t =
      format::csr_t<memory_space_t::device, vertex_t, edge_t, weight_t>;
  csr_t csr;

  // --
  // IO
  parameters_t params(num_arguments, argument_array);

  if (util::is_market(params.filename)) {
    io::matrix_market_t<vertex_t, edge_t, weight_t> mm;
    auto mmatrix = mm.load(params.filename);
    if (!mm_is_symmetric(mm.code)) {
      // std::cerr << "Error: input matrix must be symmetric" << std::endl;
      // exit(1);
    }
    csr.from_coo(mmatrix);
  } else if (util::is_binary_csr(params.filename)) {
    csr.read_binary(params.filename);
  } else {
    std::cerr << "Unknown file format: " << params.filename << std::endl;
    exit(1);
  }

  // --
  // Build graph

  auto G = graph::build::from_csr<memory_space_t::device,
                                  graph::view_t::csr>(
      csr.number_of_rows,               // rows
      csr.number_of_columns,            // columns
      csr.number_of_nonzeros,           // nonzeros
      csr.row_offsets.data().get(),     // row_offsets
      csr.column_indices.data().get(),  // column_indices
      csr.nonzero_values.data().get()   // values
  );

  // --
  // Params and memory allocation

  vertex_t n_vertices = G.get_number_of_vertices();

  using allocator_type = managed_allocator<vertex_t>;

  thrust::device_vector<vertex_t, allocator_type> mgpu_row_offsets(
      csr.row_offsets);
  thrust::device_vector<vertex_t, allocator_type> mgpu_column_indices(
      csr.column_indices);
  thrust::device_vector<vertex_t, allocator_type> output(n_vertices, 0);

  uint32_t num_gpus = 4;
  auto vertices_per_gpu = n_vertices / num_gpus;

  auto mgpu_row_offsets_raw = mgpu_row_offsets.data();
  auto mgpu_column_indices_raw = mgpu_column_indices.data();
  auto output_raw = output.data();

  cuda_try(hipDeviceSynchronize());

  // enable peer access
  for (int i = 0; i < num_gpus; i++) {
    hipSetDevice(i);
    for (int j = 0; j < num_gpus; j++) {
      if (i == j)
        continue;
      hipDeviceEnablePeerAccess(j, 0);
    }
  }

  std::cout << "Launching kernels..." << std::endl;
  float total_time = 0;
  std::size_t num_experiments = 100;
  for (std::size_t exp = 0; exp < num_experiments; exp++) {
    std::vector<std::thread> threads;

    // std::cout << "Experiment " << exp << std::endl;
    cuda_try(hipDeviceSynchronize());

    util::timer_t timer;
    timer.begin();
    for (uint32_t gpu_idx = 0; gpu_idx < num_gpus; gpu_idx++) {
      std::thread t([=] {
        // set gpu
        // std::cout << "Setting GPU " << gpu_idx << std::endl;
        cuda_try(hipSetDevice(gpu_idx));

        // offsets
        auto v_start = vertices_per_gpu * gpu_idx;
        auto v_end = std::min(vertices_per_gpu * (gpu_idx + 1), n_vertices);
        auto v_size = v_end - v_start;
        // std::cout << gpu_idx << ": " << v_start << std::endl;
        // std::cout << gpu_idx << ": " << v_end << std::endl;
        // std::cout << gpu_idx << ": " << v_size << std::endl;
        // std::cout << gpu_idx << ": " << n_vertices << std::endl;

        // launch kernel
        const uint32_t block_size = 128;
        const uint32_t num_blocks = (v_size + block_size - 1) / block_size;
        average_neighbors<<<num_blocks, block_size>>>(
            mgpu_row_offsets_raw, mgpu_column_indices_raw, n_vertices,
            output_raw, v_start, v_end);
        cuda_try(hipDeviceSynchronize());
      });
      threads.push_back(std::move(t));
    }

    // join
    for (auto& t : threads) {
      t.join();
    }
    cuda_try(hipDeviceSynchronize());
    auto elapsed = timer.end();
    total_time += elapsed;
  }

  std::cout << "Elapsed (ms): " << total_time / float(num_experiments)
            << std::endl;
}

int main(int argc, char** argv) {
  test_tc(argc, argv);
}
