#include "hip/hip_runtime.h"
#include <gunrock/algorithms/pr.hxx>

#include <nlohmann/json.hpp>

using namespace gunrock;
using namespace memory;

void test_pr(int num_arguments, char** argument_array) {
  if (num_arguments != 2) {
    std::cerr << "usage: ./bin/<program-name> filename.mtx" << std::endl;
    exit(1);
  }

  // --
  // Define types

  using vertex_t = int;
  using edge_t = int;
  using weight_t = float;

  using csr_t =
      format::csr_t<memory_space_t::device, vertex_t, edge_t, weight_t>;
  csr_t csr;

  // --
  // IO

  std::string filename = argument_array[1];

  float sort_time{0.0f};
  float convert_time{0.0f};
  vertex_t single_source = 0;  // rand() % n_vertices;

  if (util::is_market(filename)) {
    io::matrix_market_t<vertex_t, edge_t, weight_t> mm;
    auto mmatrix = mm.load(filename);
    util::timer_t sort_timer;
    sort_timer.begin();
    mmatrix.sort();
    sort_time = sort_timer.end();

    util::timer_t convert_timer;
    convert_timer.begin();
    csr.from_coo(mmatrix, single_source);
    convert_time = convert_timer.stop();
  } else if (util::is_binary_csr(filename)) {
    csr.read_binary(filename);
  } else {
    std::cerr << "Unknown file format: " << filename << std::endl;
    exit(1);
  }

  // --
  // Build graph

  auto G = graph::build::from_csr<memory_space_t::device, graph::view_t::csr>(
      csr.number_of_rows,               // rows
      csr.number_of_columns,            // columns
      csr.number_of_nonzeros,           // nonzeros
      csr.row_offsets.data().get(),     // row_offsets
      csr.column_indices.data().get(),  // column_indices
      csr.nonzero_values.data().get()   // values
  );  // supports row_indices and column_offsets (default = nullptr)

  // --
  // Params and memory allocation

  srand(time(NULL));

  weight_t alpha = 0.85;
  weight_t tol = 1e-6;

  vertex_t n_vertices = G.get_number_of_vertices();
  thrust::device_vector<weight_t> p(n_vertices);

  // --
  // GPU Run

  const int num_experiments = 10;
  double gpu_elapsed = 0.0;
  for (auto exp = 0; exp < num_experiments; exp++) {
    thrust::device_vector<weight_t> p(n_vertices);
    util::flush_cache();
    auto this_run = gunrock::pr::run(G, alpha, tol, p.data().get());
    gpu_elapsed += this_run;
  }
  gpu_elapsed /= float(num_experiments);

  using json = nlohmann::json;

  std::string app_name = "pr";
  std::string graph_name = std::filesystem::path(filename).stem();
  std::string output_dir = "pareto/";
  std::string fname =
      output_dir + app_name + std::string("_") + graph_name + ".json";
  std::fstream output(fname, std::ios::app);

  json record;
  record["graph_name"] = graph_name;
  record["M"] = G.get_number_of_edges();
  record["N"] = G.get_number_of_vertices();
  record["run-time"] = gpu_elapsed;
  record["sort-time"] = sort_time;
  record["convert-time"] = convert_time;
  output << record << "\n";
  std::cout << record << "\n";

  return;
  // --
  // Log + Validate
  // print::head(p, 40, "GPU rank");

  // std::cout << "GPU Elapsed Time : " << gpu_elapsed << " (ms)" << std::endl;
}

int main(int argc, char** argv) {
  test_pr(argc, argv);
}