#include "hip/hip_runtime.h"
#include <vector>

#include <gunrock/algorithms/tc.hxx>
#include "tc_cpu.hxx"

#include <cxxopts.hpp>

#include <nlohmann/json.hpp>

using namespace gunrock;
using namespace memory;

struct parameters_t {
  std::string filename;
  cxxopts::Options options;
  bool validate;
  bool reduce_all_triangles;

  /**
   * @brief Construct a new parameters object and parse command line arguments.
   *
   * @param argc Number of command line arguments.
   * @param argv Command line arguments.
   */
  parameters_t(int argc, char** argv)
      : options(argv[0], "Traingle Counting example") {
    // Add command line options
    options.add_options()("help", "Print help")(
        "validate", "CPU validation",
        cxxopts::value<bool>()->default_value("false"))(
        "m,market", "Matrix file", cxxopts::value<std::string>())(
        "r,reduce",
        "Compute a single triangle count for the entire graph (default = "
        "false)",
        cxxopts::value<bool>()->default_value("false"));

    // Parse command line arguments
    auto result = options.parse(argc, argv);

    if (result.count("help") || (result.count("market") == 0)) {
      std::cout << options.help({""}) << std::endl;
      std::exit(0);
    }
    filename = result["market"].as<std::string>();
    validate = result["validate"].as<bool>();
    reduce_all_triangles = result["reduce"].as<bool>();
  }
};

void test_tc(int num_arguments, char** argument_array) {
  // --
  // Define types

  using vertex_t = uint32_t;
  using edge_t = uint32_t;
  using weight_t = float;
  using count_t = vertex_t;

  using csr_t =
      format::csr_t<memory_space_t::device, vertex_t, edge_t, weight_t>;
  csr_t csr;

  // --
  // IO
  parameters_t params(num_arguments, argument_array);

  float sort_time{0.0f};
  float convert_time{0.0f};
  if (util::is_market(params.filename)) {
    io::matrix_market_t<vertex_t, edge_t, weight_t> mm;
    auto mmatrix = mm.load(params.filename);
    if (!mm_is_symmetric(mm.code)) {
      std::cerr << "Error: input matrix must be symmetric" << std::endl;
      // exit(1);
    }
    util::timer_t sort_timer;
    sort_timer.begin();
    mmatrix.sort();
    sort_time = sort_timer.end();

    util::timer_t convert_timer;
    convert_timer.begin();
    csr.from_coo(mmatrix, single_source);
    convert_time = convert_timer.stop();
  } else if (util::is_binary_csr(params.filename)) {
    csr.read_binary(params.filename);
  } else {
    std::cerr << "Unknown file format: " << params.filename << std::endl;
    exit(1);
  }

  // --
  // Build graph

  auto G = graph::build::from_csr<memory_space_t::device,
                                  graph::view_t::csr>(
      csr.number_of_rows,               // rows
      csr.number_of_columns,            // columns
      csr.number_of_nonzeros,           // nonzeros
      csr.row_offsets.data().get(),     // row_offsets
      csr.column_indices.data().get(),  // column_indices
      csr.nonzero_values.data().get()   // values
  );

  // --
  // Params and memory allocation

  vertex_t n_vertices = G.get_number_of_vertices();

  // --
  // GPU Run
  const int num_experiments = 10;
  double gpu_elapsed = 0.0;
  std::size_t total_triangles_ = 0;

  for (auto exp = 0; exp < num_experiments; exp++) {
    std::size_t total_triangles = 0;
    thrust::device_vector<count_t> triangles_count(n_vertices, 0);
    util::flush_cache();
    auto exp_gpu_elapsed =
        tc::run(G, params.reduce_all_triangles, triangles_count.data().get(),
                &total_triangles);

    total_triangles_ = total_triangles;
    gpu_elapsed += exp_gpu_elapsed;
    // std::cout << exp << " :" << exp_gpu_elapsed << std::endl;
  }
  gpu_elapsed /= double(num_experiments);

  using json = nlohmann::json;

  std::string app_name = "tc";
  std::string graph_name = std::filesystem::path(params.filename).stem();
  std::string output_dir = "pareto/";
  std::string fname =
      output_dir + app_name + std::string("_") + graph_name + ".json";
  std::fstream output(fname, std::ios::app);

  json record;
  record["graph_name"] = graph_name;
  record["M"] = G.get_number_of_edges();
  record["N"] = G.get_number_of_vertices();
  record["run-time"] = gpu_elapsed;
  record["sort-time"] = sort_time;
  record["convert-time"] = convert_time;
  record["num-triangles"] = total_triangles_;
  output << record << "\n";
  std::cout << record << "\n";

  return;
  // --
  // Log

  // print::head(triangles_count, 40, "Per-vertex triangle count");
  // if (params.reduce_all_triangles) {
  //   std::cout << "Total Graph Traingles : " << total_triangles << std::endl;
  // }
  // std::cout << "GPU Elapsed Time : " << gpu_elapsed << " (ms)" << std::endl;

  // // --
  // // CPU validation
  // if (params.validate) {
  //   std::vector<count_t> reference_triangles_count(n_vertices, 0);
  //   std::size_t reference_total_triangles = 0;

  //   float cpu_elapsed =
  //       tc_cpu::run(csr, reference_triangles_count,
  //       reference_total_triangles);
  //   uint32_t n_errors = 0;
  //   if (total_triangles != reference_total_triangles) {
  //     std::cout << "Error: Total TC mismatch: " << total_triangles
  //               << "! = " << reference_total_triangles << std::endl;
  //     n_errors++;
  //   }
  //   n_errors += util::compare(
  //       triangles_count.data().get(), reference_triangles_count.data(),
  //       n_vertices, [](const auto x, const auto y) { return x != y; }, true);
  //   std::cout << "CPU Elapsed Time : " << cpu_elapsed << " (ms)" <<
  //   std::endl; std::cout << "Number of errors : " << n_errors << std::endl;
  // }
}

int main(int argc, char** argv) {
  test_tc(argc, argv);
}
